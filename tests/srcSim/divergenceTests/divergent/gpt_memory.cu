#include <iostream>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 256
#define ARRAY_SIZE 1024

__global__ void uncoalescedMemoryAccessKernel(int *d_in, int *d_out, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < n) {
        int offset = (idx % 4) * 256;
        d_out[idx] = d_in[offset];
    }
}

void uncoalescedMemoryAccess(int *h_in, int *h_out, int n) {
    int *d_in, *d_out;
    size_t bytes = n * sizeof(int);

    hipMalloc(&d_in, bytes);
    hipMalloc(&d_out, bytes);

    hipMemcpy(d_in, h_in, bytes, hipMemcpyHostToDevice);

    int blockSize = BLOCK_SIZE;
    int gridSize = (n + blockSize - 1) / blockSize;

    uncoalescedMemoryAccessKernel<<<gridSize, blockSize>>>(d_in, d_out, n);

    hipMemcpy(h_out, d_out, bytes, hipMemcpyDeviceToHost);

    hipFree(d_in);
    hipFree(d_out);
}

int main() {
    const int N = ARRAY_SIZE;
    int h_in[N], h_out[N];

    for (int i = 0; i < N; i++) {
        h_in[i] = i;
    }

    uncoalescedMemoryAccess(h_in, h_out, N);

    for (int i = 0; i < N; i++) {
        std::cout << h_out[i] << " ";
    }
    std::cout << std::endl;

    return 0;
}
