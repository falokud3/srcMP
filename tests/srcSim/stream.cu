
#include <hip/hip_runtime.h>
// Source: https://github.com/bcumming/cuda-stream/blob/master/stream.cu


template <typename T>
__global__ void STREAM_Copy(T const * __restrict__ const a, T * __restrict__ const b, int len)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < len)
        b[idx] = a[idx];
}

template <typename T>
__global__ void STREAM_Scale(T const * __restrict__ const a, T * __restrict__ const b, T scale,  int len)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < len)
        b[idx] = scale * a[idx];
}

template <typename T>
__global__ void STREAM_Add(T const * __restrict__ const a, T const * __restrict__ const b, T * __restrict__ const c, int len)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < len)
        c[idx] = a[idx] + b[idx];
}

template <typename T>
__global__ void STREAM_Triad(T const * __restrict__ a, T const * __restrict__ b, T * __restrict__ const c, T scalar, int len)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < len)
        c[idx] = a[idx] + scalar * b[idx];
}